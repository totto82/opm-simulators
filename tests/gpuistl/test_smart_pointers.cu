#include "hip/hip_runtime.h"
/*
  Copyright 2025 Equinor ASA

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestSmartPointers

#include <boost/test/unit_test.hpp>
#include <opm/simulators/linalg/gpuistl/gpu_smart_pointer.hpp>

namespace
{

struct SomeStruct {
    __device__ void someFunction()
    {
        this->isCalled = true;
    }

    bool isCalled = false;
};

template <class T>
__global__ void
setValue(Opm::gpuistl::PointerView<T> ptrIn, Opm::gpuistl::PointerView<T> ptrOut)
{
    *ptrOut = *ptrIn;
}

template <class T>
__global__ void
setValueGet(Opm::gpuistl::PointerView<T> ptrIn, Opm::gpuistl::PointerView<T> ptrOut)
{
    *ptrOut.get() = *ptrIn.get();
}

template <class T>
__global__ void
callFunction(Opm::gpuistl::PointerView<T> ptrIn)
{
    ptrIn->someFunction();
}


} // namespace


BOOST_AUTO_TEST_CASE(TestSharedPointer)
{
    auto sharedPtr = Opm::gpuistl::make_gpu_shared_ptr<int>(1);

    int valueFromDevice = 0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&valueFromDevice, sharedPtr.get(), sizeof(int), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(valueFromDevice, 1);
}

BOOST_AUTO_TEST_CASE(TestUniquePointer)
{
    auto uniquePtr = Opm::gpuistl::make_gpu_unique_ptr<int>(1);

    int valueFromDevice = 0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&valueFromDevice, uniquePtr.get(), sizeof(int), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(valueFromDevice, 1);
}

BOOST_AUTO_TEST_CASE(TestPointerView)
{
    auto pointerDestination = Opm::gpuistl::make_gpu_shared_ptr<double>(92);
    auto pointerSource = Opm::gpuistl::make_gpu_shared_ptr<double>(128.5);

    setValue<<<1, 1>>>(Opm::gpuistl::make_view(pointerSource), Opm::gpuistl::make_view(pointerDestination));

    double valueFromDevice = 0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&valueFromDevice, pointerDestination.get(), sizeof(double), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(valueFromDevice, 128.5);

    auto newSource = Opm::gpuistl::make_gpu_shared_ptr<double>(-1.0);
    setValueGet<<<1, 1>>>(Opm::gpuistl::make_view(newSource), Opm::gpuistl::make_view(pointerDestination));
    OPM_GPU_SAFE_CALL(hipMemcpy(&valueFromDevice, pointerDestination.get(), sizeof(double), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(valueFromDevice, -1.0);

    auto structPtr = Opm::gpuistl::make_gpu_shared_ptr<SomeStruct>();
    callFunction<<<1, 1>>>(Opm::gpuistl::make_view(structPtr));
    bool isCalled = false;
    OPM_GPU_SAFE_CALL(hipMemcpy(&isCalled, structPtr.get(), sizeof(bool), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(isCalled, true);


    auto uniquePtr = Opm::gpuistl::make_gpu_unique_ptr<double>(1.0);
    auto uniqueView = Opm::gpuistl::make_view(uniquePtr);

    double valueFromDeviceUnique = 0;
    OPM_GPU_SAFE_CALL(hipMemcpy(&valueFromDeviceUnique, uniqueView.get(), sizeof(double), hipMemcpyDeviceToHost));
    BOOST_CHECK_EQUAL(valueFromDeviceUnique, 1.0);
}
